#include "hip/hip_runtime.h"
// int xp = y*pars.Nx*pars.Nz + (x+1)*pars.Nz + z;
// int xm = y*pars.Nx*pars.Nz + (x-1)*pars.Nz + z;
// int zp = y*pars.Nx*pars.Nz + x*pars.Nz + z + 1;
// int zm = y*pars.Nx*pars.Nz + x*pars.Nz + z - 1;
// int yp = (y+1)*pars.Nx*pars.Nz + x*pars.Nz + z;
// int ym = (y-1)*pars.Nx*pars.Nz + x*pars.Nz + z;
#include "params.cuh"

extern __constant__ Params pars;

__global__ void calc_fdtd_step_x(
    ftype* e, 
    ftype* h1, 
    ftype* h2,
    ftype* ca,
    ftype* cb
) {
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    int z = blockIdx.z*blockDim.z + threadIdx.z;
    int c = y*pars.Nx*pars.Nz + x*pars.Nz + z;
    if (x < pars.Nx && y < pars.Ny && z < pars.Nz) {
        int zp = y*pars.Nx*pars.Nz + x*pars.Nz + z + 1;
        int yp = (y+1)*pars.Nx*pars.Nz + x*pars.Nz + z;
        e[c] = ca[c] * e[c] + cb[c] * (
            h1[yp] - h1[c] - h2[zp] + h2[c]
        );
    }
}
__global__ void calc_fdtd_step_y(
    ftype* e, 
    ftype* h1, 
    ftype* h2,
    ftype* j,
    ftype* ca,
    ftype* cb
) {
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    int z = blockIdx.z*blockDim.z + threadIdx.z;
    int c = y*pars.Nx*pars.Nz + x*pars.Nz + z;
    if (x < pars.Nx && y < pars.Ny && z < pars.Nz) {
        int zp = y*pars.Nx*pars.Nz + x*pars.Nz + z + 1;
        int xp = y*pars.Nx*pars.Nz + (x+1)*pars.Nz + z;
        e[c] = ca[c] * e[c] + cb[c] * (
            h1[zp] - h1[c] - h2[xp] + h2[c] - j[c] * pars.dr
        );
    }
}
__global__ void calc_fdtd_step_z(
    ftype* e, 
    ftype* h1, 
    ftype* h2,
    ftype* j,
    ftype* ca,
    ftype* cb
) {
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    int z = blockIdx.z*blockDim.z + threadIdx.z;
    int c = y*pars.Nx*pars.Nz + x*pars.Nz + z;
    if (x < pars.Nx && y < pars.Ny && z < pars.Nz) {
        int xp = y*pars.Nx*pars.Nz + (x+1)*pars.Nz + z;
        int yp = (y+1)*pars.Nx*pars.Nz + x*pars.Nz + z;
        e[c] = ca[c] * e[c] + cb[c] * (
            h1[yp] - h1[c] - h2[xp] + h2[c] - j[c] * pars.dr
        );
    }
}

__global__ void calc_ca(
    ftype* ca,
    ftype* sigma,
    ftype* epsilon
) {
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    int z = blockIdx.z*blockDim.z + threadIdx.z;
    int c = y*pars.Nx*pars.Nz + x*pars.Nz + z;
    if (x < pars.Nx && y < pars.Ny && z < pars.Nz) {
        ca[c] = (1 - sigma[c] * pars.dt / (2*epsilon[c])) / 
        (1 + sigma[c] * pars.dt / (2*epsilon[c]));
    }
}

__global__ void calc_cb(
    ftype* cb,
    ftype* sigma,
    ftype* epsilon
) {
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    int z = blockIdx.z*blockDim.z + threadIdx.z;
    int c = y*pars.Nx*pars.Nz + x*pars.Nz + z;
    if (x < pars.Nx && y < pars.Ny && z < pars.Nz) {
        cb[c] = (pars.dt / (2*epsilon[c]*pars.dr)) / 
        (1 + sigma[c] * pars.dt / (2*epsilon[c]));
    }
}

__global__ void calc_fdtd_step_2d_x(
    ftype* field1,
    ftype* field2z,
    ftype* perm,
    Offset off
) {
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    if (x < pars.Nx-1 && y < pars.Ny-1 && x > 0 && y > 0) {
        int c = x*pars.Ny + y;
        int left  = (x + off.lx)*pars.Ny + y + off.ly;
        int right = (x + off.rx)*pars.Ny + y + off.ry;
        field1[c] += pars.c * pars.dt * (field2z[left] - field2z[right]) / (pars.dr * perm[c]);
    }
}

__global__ void calc_fdtd_step_2d_y(
    ftype* field1,
    ftype* field2z,
    ftype* perm,
    Offset off
) {
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    if (x < pars.Nx-1 && y < pars.Ny-1 && x > 0 && y > 0) {
        int c = x*pars.Ny + y;
        int left  = (x + off.lx)*pars.Ny + y + off.ly;
        int right = (x + off.rx)*pars.Ny + y + off.ry;
        field1[c] += - pars.c * pars.dt * (field2z[left] - field2z[right]) / (pars.dr * perm[c]);
    }
}

__global__ void calc_fdtd_step_2d_z(
    ftype* field1,
    ftype* field2y,
    ftype* field2x,
    ftype* perm,
    Offset off1,
    Offset off2
) {
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    if (x < pars.Nx-1 && y < pars.Ny-1 && x > 0 && y > 0) {
        int c = x*pars.Ny + y;
        int left1  = (x + off1.lx)*pars.Ny + y + off1.ly;
        int right1 = (x + off1.rx)*pars.Ny + y + off1.ry;
        int left2  = (x + off2.lx)*pars.Ny + y + off2.ly;
        int right2 = (x + off2.rx)*pars.Ny + y + off2.ry;
        field1[c] += - pars.c * pars.dt * (field2y[left1] - field2y[right1] - field2x[left2] + field2x[right2]) / (pars.dr * perm[c]);
    }
}

__global__ void inject_soft_source_2d(
    ftype* field,
    ftype value
) {
    int c = pars.source_x*pars.Ny + pars.source_y;
    field[c] += value;
}