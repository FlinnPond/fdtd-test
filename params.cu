#include "params.cuh"

__host__ void check_err(hipError_t err, const char* step_name)
 {
    if (err != hipSuccess) {
        fprintf(stderr, "Error during %s: %s.\n", step_name, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}
void Params::init_pars() {
    dr = 1e-6;
    dt = 1e-8;
    Nx = 300;
    Ny = 300;
    Nz = 300;
    eps_0 = 8.854e-12;
    mu_0 = 1.257e-6;
    n_steps = 1000;

    drop_rate = 100;
    
    c = 3e8;

    xm.lx=0; xm.ly=0; xm.lz=0; xm.rx=-1;xm.ry=0; xm.rz=0;
    ym.lx=0; ym.ly=0; ym.lz=0; ym.rx=0; ym.ry=-1;ym.rz=0;
    zm.lx=0; zm.ly=0; zm.lz=0; zm.rx=0; zm.ry=0; zm.rz=-1;
    xp.lx=1; xp.ly=0; xp.lz=0; xp.rx=0; xp.ry=0; xp.rz=0;
    yp.lx=0; yp.ly=1; yp.lz=0; yp.rx=0; yp.ry=0; yp.rz=0;
    zp.lx=0; zp.ly=0; zp.lz=1; zp.rx=0; zp.ry=0; zp.rz=0;
}
void Params::init_memory_2d() {
    host.ex = (ftype*)(malloc(Nx*Ny*sizeof(ftype)));
    host.ey = (ftype*)(malloc(Nx*Ny*sizeof(ftype)));
    host.ez = (ftype*)(malloc(Nx*Ny*sizeof(ftype)));
    host.hx = (ftype*)(malloc(Nx*Ny*sizeof(ftype)));
    host.hy = (ftype*)(malloc(Nx*Ny*sizeof(ftype)));
    host.hz = (ftype*)(malloc(Nx*Ny*sizeof(ftype)));
    host.mu = (ftype*)(malloc(Nx*Ny*sizeof(ftype)));
    host.eps= (ftype*)(malloc(Nx*Ny*sizeof(ftype)));

    check_err(hipMalloc(reinterpret_cast<void **>(&device.ex), Nx*Ny*sizeof(ftype)), "allocating");
    check_err(hipMalloc(reinterpret_cast<void **>(&device.ey), Nx*Ny*sizeof(ftype)), "allocating");
    check_err(hipMalloc(reinterpret_cast<void **>(&device.ez), Nx*Ny*sizeof(ftype)), "allocating");
    check_err(hipMalloc(reinterpret_cast<void **>(&device.hx), Nx*Ny*sizeof(ftype)), "allocating");
    check_err(hipMalloc(reinterpret_cast<void **>(&device.hy), Nx*Ny*sizeof(ftype)), "allocating");
    check_err(hipMalloc(reinterpret_cast<void **>(&device.hz), Nx*Ny*sizeof(ftype)), "allocating");
    check_err(hipMalloc(reinterpret_cast<void **>(&device.mu), Nx*Ny*sizeof(ftype)), "allocating");
    check_err(hipMalloc(reinterpret_cast<void **>(&device.eps),Nx*Ny*sizeof(ftype)), "allocating");

    for (int x = 0; x < Nx; x++){
        for (int y = 0; y < Ny; y++) {
            int c = x * Ny + y;
            host.ex[c] = 0;
            host.ey[c] = 0;
            host.ez[c] = 0;
            host.hx[c] = 0;
            host.hy[c] = 0;
            host.hz[c] = 0;
            host.mu[c] = 1;
            host.eps[c]= 1;
        }
    }

    check_err(hipMemcpy(device.ex, host.ex, Nx*Ny*sizeof(ftype), hipMemcpyHostToDevice), "copying to device");
    check_err(hipMemcpy(device.ey, host.ey, Nx*Ny*sizeof(ftype), hipMemcpyHostToDevice), "copying to device");
    check_err(hipMemcpy(device.ez, host.ez, Nx*Ny*sizeof(ftype), hipMemcpyHostToDevice), "copying to device");
    check_err(hipMemcpy(device.hx, host.hx, Nx*Ny*sizeof(ftype), hipMemcpyHostToDevice), "copying to device");
    check_err(hipMemcpy(device.hy, host.hy, Nx*Ny*sizeof(ftype), hipMemcpyHostToDevice), "copying to device");
    check_err(hipMemcpy(device.hz, host.hz, Nx*Ny*sizeof(ftype), hipMemcpyHostToDevice), "copying to device");
    check_err(hipMemcpy(device.mu, host.mu, Nx*Ny*sizeof(ftype), hipMemcpyHostToDevice), "copying to device");
    check_err(hipMemcpy(device.eps,host.eps,Nx*Ny*sizeof(ftype), hipMemcpyHostToDevice), "copying to device");
}
void Params::extract_data_2d(){
    hipMemcpy(host.ex,device.ex,Nx*Ny*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(host.ey,device.ey,Nx*Ny*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(host.ez,device.ez,Nx*Ny*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(host.hx,device.hx,Nx*Ny*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(host.hy,device.hy,Nx*Ny*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(host.hz,device.hz,Nx*Ny*sizeof(double), hipMemcpyDeviceToHost);
}
void Params::free_memory() {
    free(host.ex);
    free(host.ey);
    free(host.ez);
    free(host.hx);
    free(host.hy);
    free(host.hz);
    free(host.mu);
    free(host.eps);
    check_err(hipFree(device.ex), "cleaning");
    check_err(hipFree(device.ey), "cleaning");
    check_err(hipFree(device.ez), "cleaning");
    check_err(hipFree(device.hx), "cleaning");
    check_err(hipFree(device.hy), "cleaning");
    check_err(hipFree(device.hz), "cleaning");
    check_err(hipFree(device.mu), "cleaning");
    check_err(hipFree(device.eps),"cleaning");
}
