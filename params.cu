#include "params.cuh"

__host__ void check_err(hipError_t err, const char* step_name)
 {
    if (err != hipSuccess) {
        fprintf(stderr, "Error during %s: %s.\n", step_name, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}
